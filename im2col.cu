#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "cuda_common.hpp"

#include "conv/ops/im2col/im2col.hpp"

#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cutlass/gemm/device/gemm.h>
#include "cutlass/gemm/device/gemm_splitk_parallel.h"

#include <hipblas.h>

constexpr int N = 1, C = 512, H = 108, W = 108;
constexpr int K = 128, M = 3, D = 1, S = 1;
constexpr int G = 1;

constexpr int P = M / 2;

constexpr bool USE_CUTLASS = true;
constexpr bool USE_SPLITK = false;

const auto MAP_H = (H + 2 * P - ((M - 1) * D + 1)) / S + 1;
const auto MAP_W = (W + 2 * P - ((M - 1) * D + 1)) / S + 1;

constexpr int TRIALS = 1;

float *cudnn_workspacePtr = nullptr;
float *mine_workspacePtr = nullptr;

float benchmark_cudnn(const float* inputPtr, const float* filterPtr, float* outputPtr)
{
    hipdnnHandle_t handle;
    CHECK_CUDNN(hipdnnCreate(&handle));
    
    hipdnnTensorDescriptor_t inputDesc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&inputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

    hipdnnFilterDescriptor_t filterDesc;
    CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
    CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, K, C, M, M));

    hipdnnConvolutionDescriptor_t convDesc;
    CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
    CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(convDesc, P, P, S, S, D, D, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    CHECK_CUDNN(hipdnnSetConvolutionGroupCount(convDesc, G));

    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;

    hipdnnTensorDescriptor_t outputDesc;
    CHECK_CUDNN(hipdnnCreateTensorDescriptor(&outputDesc));
    CHECK_CUDNN(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, K, MAP_H, MAP_W));

    size_t workspaceSize = 0;
    CHECK_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(handle, inputDesc, filterDesc, convDesc, outputDesc, algo, &workspaceSize));
    std::cout << "workspace size: "  << workspaceSize / 1024 / 1024 << "MB" << std::endl;

    float *workspacePtr = nullptr;
    CHECK_CUDA(hipMalloc(&workspacePtr, workspaceSize));
    cudnn_workspacePtr = workspacePtr;

    float time = 0;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    {
        for (int i = 0; i < TRIALS; i++)
        {
            CHECK_CUDA(hipEventRecord(start));
            float alpha = 1.0, beta = 0.0;
            CHECK_CUDNN(hipdnnConvolutionForward(handle, &alpha, inputDesc, inputPtr, filterDesc, filterPtr, convDesc, algo, workspacePtr, workspaceSize, &beta, outputDesc, outputPtr));
            CHECK_CUDA(hipEventRecord(stop));

            CHECK_CUDA(hipEventSynchronize(stop));

            float cur_time;
            CHECK_CUDA(hipEventElapsedTime(&cur_time, start, stop));
            time += cur_time;
        }

        time /= TRIALS;
    }
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    //CHECK_CUDA(hipFree(workspacePtr));
    
    return time;
}

template <class T>
struct IdentityFunctor
{
    CUTLASS_HOST_DEVICE IdentityFunctor() { }
    CUTLASS_HOST_DEVICE T operator()(T value) const { return  value; }
};

template <class T>
struct SigmoidFunctor
{
    CUTLASS_HOST_DEVICE SigmoidFunctor() { }
    CUTLASS_HOST_DEVICE T operator()(T value) const { return  1.0 / (1.0 + __expf(-value)); }
};

// Epilogue copied and modified from cutlass::LinearCombinationRelu
template <
    typename ElementOutput_,
    typename ActivationFunctor = IdentityFunctor<ElementOutput_>,
    int Count = 1,
    typename ElementAccumulator_ = ElementOutput_,
    typename ElementCompute_ = ElementOutput_,
    cutlass::FloatRoundStyle Round = cutlass::FloatRoundStyle::round_to_nearest
>
class BaseEpilogue
{
public:
    using ElementOutput = ElementOutput_;
    using ElementAccumulator = ElementAccumulator_;
    using ElementCompute = ElementCompute_;

    static const int kCount = Count;

    using FragmentOutput = cutlass::Array<ElementOutput, kCount>;
    using FragmentAccumulator = cutlass::Array<ElementAccumulator, kCount>;
    using ComputeFragment = cutlass::Array<ElementCompute, kCount>;

    static const auto kRound = Round;

    struct Params
    {
        ElementCompute alpha, beta;
        ElementCompute const *alpha_ptr, *beta_ptr;

        ActivationFunctor functor;

        CUTLASS_HOST_DEVICE
        Params()
            : alpha(ElementCompute(1)), beta(ElementCompute(0)), alpha_ptr(nullptr), beta_ptr(nullptr) { }

        CUTLASS_HOST_DEVICE
        Params(ElementCompute alpha, ElementCompute beta)
            : alpha(alpha), beta(beta), alpha_ptr(nullptr), beta_ptr(nullptr) { }

        CUTLASS_HOST_DEVICE
        Params(ElementCompute alpha, ElementCompute beta, ActivationFunctor functor)
            : alpha(alpha), beta(beta), alpha_ptr(nullptr), beta_ptr(nullptr), functor(functor) { }

        CUTLASS_HOST_DEVICE
        Params(ElementCompute const *alpha_ptr, ElementCompute const *beta_ptr)
            : alpha(0), beta(0), alpha_ptr(alpha_ptr), beta_ptr(beta_ptr) { }

        CUTLASS_HOST_DEVICE
        Params(ElementCompute const *alpha_ptr, ElementCompute const *beta_ptr, ActivationFunctor functor)
            : alpha(0), beta(0), alpha_ptr(alpha_ptr), beta_ptr(beta_ptr), functor(functor) { }
    };

private:
    ElementCompute alpha_, beta_;
    ActivationFunctor functor;

public:
    CUTLASS_HOST_DEVICE
    BaseEpilogue(Params const &params)
    {
        alpha_ = (params.alpha_ptr ? *params.alpha_ptr : params.alpha);
        beta_ = (params.beta_ptr ? *params.beta_ptr : params.beta);
        functor = params.functor;
    }

    CUTLASS_HOST_DEVICE
    bool is_source_needed() const
    {
        return beta_ != ElementCompute(0);
    }

    CUTLASS_HOST_DEVICE
    void set_k_partition(int k_partition)
    {
        if (k_partition)
            beta_ = ElementCompute(1);
    }

    CUTLASS_HOST_DEVICE
    FragmentOutput operator()(
        FragmentAccumulator const &accumulator, 
        FragmentOutput const &source,
        ElementCompute uniform = 0) const
    {
        cutlass::NumericArrayConverter<ElementCompute, ElementOutput, kCount, Round> source_converter;
        cutlass::NumericArrayConverter<ElementCompute, ElementAccumulator, kCount, Round> accumulator_converter;

        ComputeFragment converted_source = source_converter(source);
        ComputeFragment converted_accumulator = accumulator_converter(accumulator);

        ComputeFragment intermediate;
        cutlass::multiplies<ComputeFragment> mul_add_source;
        cutlass::multiply_add<ComputeFragment> mul_add_accumulator;
    
        intermediate = mul_add_source(beta_, converted_source);
        intermediate = mul_add_accumulator(alpha_, converted_accumulator, intermediate);
        
        for (int i = 0; i < ComputeFragment::kElements; i++)
            intermediate[i] = functor(intermediate[i]);

        cutlass::NumericArrayConverter<ElementOutput, ElementCompute, kCount, Round> destination_converter;
        return destination_converter(intermediate);
    }
};

float benchmark_mine(const float* inputPtr, const float* filterPtr, float* outputPtr)
{
    size_t workspaceSize = (M * M * C) * (MAP_H * MAP_W) * sizeof(float);
    std::cout << "workspace size: "  << workspaceSize / 1024 / 1024 << "MB" << std::endl;

    float *workspacePtr = nullptr;
    CHECK_CUDA(hipMalloc(&workspacePtr, workspaceSize));
    mine_workspacePtr = workspacePtr;

    ops::im2col::Im2col<float, float> op;
    op.set_configuration({N, C, H, W}, // input shape
                         {K, C, M, M}, // filter shape
                         {S, S}, // stride
                         {D, D}, // dilation
                         {P, P}, // lpadding
                         {P, P} // rpadding
                     );

    // op.autotune(workspacePtr, inputPtr, 0, 20);

    // std::cout << "Im2col Op: " << op.save() << std::endl;
    // op.load(op.save());

    op.load("im2col_2d_kd 3 3 1 1 1 1 512");
    std::cout << "Im2col Op: " << op.save() << std::endl;

    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    using Gemm = cutlass::gemm::device::Gemm<
        float, cutlass::layout::RowMajor, // A
        float, cutlass::layout::RowMajor, // B
        float, cutlass::layout::RowMajor, // C
        float,                            // Accumulator
        cutlass::arch::OpClassSimt,       // Operation type
        cutlass::arch::Sm61,               // target arch
        cutlass::gemm::GemmShape<32, 128, 8>, // threadblock shape
        cutlass::gemm::GemmShape<32, 64, 8>, // warp shape
        cutlass::gemm::GemmShape<1, 1, 1>, // instruction shape
        BaseEpilogue<float, IdentityFunctor<float>>
    >;

    using GemmSplitK = cutlass::gemm::device::GemmSplitKParallel<
        float, cutlass::layout::RowMajor, // A
        float, cutlass::layout::RowMajor, // B
        float, cutlass::layout::RowMajor, // C
        float,                            // Accumulator
        cutlass::arch::OpClassSimt,       // Operation type
        cutlass::arch::Sm61,               // target arch
        cutlass::gemm::GemmShape<32, 64, 8>, // threadblock shape
        cutlass::gemm::GemmShape<8, 64, 8>, // warp shape
        cutlass::gemm::GemmShape<1, 1, 1>, // instruction shape
        BaseEpilogue<float, IdentityFunctor<float>>>;

    Gemm gemm_op;
    cutlass::Status status;

    GemmSplitK gemm_splitK_op;

    if (USE_SPLITK)
    {
        typename GemmSplitK::Arguments arguments{
            {K, MAP_H * MAP_W, M * M * C},
            {filterPtr, M * M * C},
            {workspacePtr, MAP_H * MAP_W},
            {outputPtr, MAP_H * MAP_W},
            {outputPtr, MAP_H * MAP_W},
            {1.0, 0.0},
            32
        };
    
        size_t workspace_size = GemmSplitK::get_workspace_size(arguments);
    
        void* gemm_workspace = nullptr;
        CHECK_CUDA(hipMalloc(&gemm_workspace, workspace_size));
    
        gemm_splitK_op.initialize(arguments, gemm_workspace);
    }

    float time = 0;
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    {
        for (int i = 0; i < TRIALS; i++)
        {
            CHECK_CUDA(hipEventRecord(start));

            op.run(workspacePtr, inputPtr);

            if (USE_CUTLASS)
            {
                if (USE_SPLITK)
                {
                    status = gemm_splitK_op();
                }
                else
                {
                    status = gemm_op({
                        {K, MAP_H * MAP_W, M * M * C},
                        {filterPtr, M * M * C},
                        {workspacePtr, MAP_H * MAP_W},
                        {outputPtr, MAP_H * MAP_W},
                        {outputPtr, MAP_H * MAP_W},
                        {1.0, 0.0}          // epilogue operation arguments
                    });
                }                
            }
            else
            {
                float alpha = 1.0, beta = 0.0;
                CHECK_CUBLAS(hipblasSgemm(handle,
                    HIPBLAS_OP_N, HIPBLAS_OP_N,
                    H * W, K, M * M * C,
                    &alpha,
                    workspacePtr, H * W,
                    filterPtr, M * M * C,                
                    &beta,
                    outputPtr, H * W
                ));
            }

            CHECK_CUDA(hipEventRecord(stop));
            CHECK_CUDA(hipEventSynchronize(stop));

            float cur_time;
            CHECK_CUDA(hipEventElapsedTime(&cur_time, start, stop));
            time += cur_time;
        }

        time /= TRIALS;
    }
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    return time;
}

int main ()
{
    float *input = nullptr;
    {
        CHECK_CUDA(hipMalloc(&input, N * C * H * W * sizeof(float)));

        float *input_h = new float[N * C * H * W];
        for (int i = 0; i < N * C * H * W; i++)
            input_h[i] = (i % 1024)/ 1024.0;
        CHECK_CUDA(hipMemcpy(input, input_h, N * C * H * W * sizeof(float), hipMemcpyHostToDevice));
    }

    float *filters = nullptr;
    {
        CHECK_CUDA(hipMalloc(&filters, K * C * M * M * sizeof(float)));

        float *filters_h = new float[K * C * M * M];
        for (int i = 0; i < K * C * M * M; i++)
        filters_h[i] = (i % 128) / 128.0;
        CHECK_CUDA(hipMemcpy(filters, filters_h, K * C * M * M * sizeof(float), hipMemcpyHostToDevice));
    }

    constexpr int output_size = N * K * MAP_H * MAP_W;

    float *output_cudnn = nullptr;
    CHECK_CUDA(hipMalloc(&output_cudnn, output_size * sizeof(float)));

    auto cudnn_time = benchmark_cudnn(input, filters, output_cudnn);
    std::cout << "cuDNN time: " << cudnn_time  << std::endl;

    float *output_mine = nullptr;
    CHECK_CUDA(hipMalloc(&output_mine, output_size * sizeof(float)));

    auto mine_time = benchmark_mine(input, filters, output_mine);
    std::cout << "My time: " << mine_time  << std::endl;

    constexpr int im2col_size = M * M * C * MAP_H * MAP_W;
    float *im2col_cudnn = new float[im2col_size];
    float *im2col_mine = new float[im2col_size];
    CHECK_CUDA(hipMemcpy(im2col_cudnn, cudnn_workspacePtr, im2col_size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(im2col_mine, mine_workspacePtr, im2col_size * sizeof(float), hipMemcpyDeviceToHost));

    double im2col_err_norm = 0.0;
    for (int i = 0; i < im2col_size; i++)
    {
        auto diff = (im2col_cudnn[i] - im2col_mine[i]);
        im2col_err_norm += diff * diff;
    }
    std::cout << "im2col L2 error norm: " << ' ' << std::sqrt(im2col_err_norm / im2col_size) << std::endl;

    float *output_cudnn_h = new float[output_size];
    float *output_mine_h = new float[output_size];
    CHECK_CUDA(hipMemcpy(output_cudnn_h, output_cudnn, output_size * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(output_mine_h, output_mine, output_size * sizeof(float), hipMemcpyDeviceToHost));

    double conv_err_norm = 0.0;
    for (int i = 0; i < output_size; i++)
    {
        auto diff = (output_cudnn_h[i] - output_mine_h[i]);
        conv_err_norm += diff * diff;
    }
    std::cout << "Conv L2 error norm: " << ' ' << std::sqrt(conv_err_norm / output_size) << std::endl;
    return 0;
}